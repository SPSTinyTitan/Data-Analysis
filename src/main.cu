
#include <cstdlib>
#include <iostream>   
#include <iomanip>
#include <cmath>

#include "vector_wrappers.h"
#include "matrix_wrappers.h"
#include "fitting.h"
#include "cuda_tools.h"



//References:
//https://devblogs.nvidia.com/using-shared-memory-cuda-cc/
//https://docs.nvidia.com/cuda/cublas/index.html
//https://docs.nvidia.com/cuda/cusolver/index.html


namespace fit{

    __host__ void pInv(float* A, int M, int N){

        int P = M;
        if (M > N) P = N;

        float* S;       gpuErrchk(hipMalloc(&S, M * sizeof(float)));
        float* U;       gpuErrchk(hipMalloc(&U, M * M * sizeof(float)));
        float* VT;      gpuErrchk(hipMalloc(&VT, N * N * sizeof(float)));
        float* W;       gpuErrchk(hipMalloc(&W, M * N * sizeof(float)));
        gpuErrchk(hipMemset(W, 0, M * N * sizeof(float)));
        gpuErrchk(hipMemset(S, 0, M * sizeof(float)));
        svd(A, M, N, S, U, VT);

        matrix::multD(S, VT, W, N, M);
        matrix::mult(U, W, A, 1, 1, false, false, M, N, N);

        vector::div(1,S,S,P);
        matrix::multD(S, VT, W, N, M);
        matrix::mult(U, W, A, 1, 1, false, false, M, N, N);
        matrix::transpose(A, A, M, N);

        if (S)  hipFree(S);
        if (U)  hipFree(U);
        if (VT) hipFree(VT);
        if (W)  hipFree(W);

    }

    //Gauss-Newton method
    __host__ void FitGaussNewton(float* A, float* param, void f(float* A, float* param, int N), int N, int K){

        //Finding inverse of Jacobian
        float* J;       gpuErrchk(hipMalloc(&J, N * K * sizeof(float)));
        float* F;       gpuErrchk(hipMalloc(&F, N * sizeof(float)));
        float* P;       gpuErrchk(hipMalloc(&P, K * sizeof(float)));
        float* P_h =    (float*)malloc(K *sizeof(float));
        
        for (int j = 0; j < 10; j++){
        //J^-1
        jacobian_v2(J, f, param, N, K);
        pInv(J, N, K);
        
        //y - f(param)
        fit::doubleExp(F, param, N);
        vector::sub(A, F, F, N);

        //J^-1(y - f(param))
        matrix::mult(J, F, P, 1, 1, false, false, K, N, 1);

        //Update param
        hipMemcpy(P_h, P, K * sizeof(float), hipMemcpyDeviceToHost);
        for (int i = 0; i < K; i++){
            param[i] += 0.1 * P_h[i];
        }
        
        fit::doubleExp(F, param, N);
        vector::sub(A, F, F, N);
        std::cout << "Error:" << vector::sum(F, N) << '\n';
        }
        
        if (J)  hipFree(J);
        if (F)  hipFree(F);
    }

}

void TestMult(){
    int M = 4;
    int N = 4;
    int K = 3;
    float A[16] = {3, 9, 2, 1, 1, 3, 5, 0, 3, 9, 2, 1, 1, 3, 5, 0};
    float B[12] = {2, 1, 2, 8, 9, 3, 4 ,1, 0, 5, 7, 5};
    float* A_d; gpuErrchk(hipMalloc(&A_d, M * N * sizeof(float)));
    float* B_d; gpuErrchk(hipMalloc(&B_d, N * K * sizeof(float)));
    float* C_d; gpuErrchk(hipMalloc(&C_d, M * K * sizeof(float)));
    hipMemcpy(A_d, &A, M * N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(B_d, &B, N * K * sizeof(float), hipMemcpyHostToDevice);
    matrix::mult(A_d, B_d, C_d, 1, 1, false, false, M, N, K);
    print_(C_d, M, K);
    std::cout << "\nTranspose: \n";
    matrix::transpose(C_d, C_d, M, K);
    print_(C_d, K, M);

    //Expected Output:

    // 21             43             31             
    // 63             129            93             
    // 53             46             64             
    // 4              13             7              
    
    // Transpose: 
    // 21             63             53             4              
    // 43             129            46             13             
    // 31             93             64             7   
}

void TestSvd(){
    int M = 3;
    int N = 2;

    int P = M;
    if (M > N)
        P = N;

    float* result_h = (float*)malloc(M * N * sizeof(float)); 
    float* S;       gpuErrchk(hipMalloc(&S, P * sizeof(float)));
    float* U;       gpuErrchk(hipMalloc(&U, M * M * sizeof(float)));
    float* VT;      gpuErrchk(hipMalloc(&VT, N * N * sizeof(float)));
    float* W;       gpuErrchk(hipMalloc(&W, M * N * sizeof(float)));

    float A_h[20] = {
        2,0,0,
        0,-3,0};
    float* A_d;
    gpuErrchk(hipMalloc(&A_d, M * N * sizeof(float)));
    hipMemcpy(A_d, &A_h, M * N * sizeof(float), hipMemcpyHostToDevice);

    fit::svd(A_d, M, N, S, U, VT);
    
    std::cout << "\nU:\n";    print_(U, M, M);
    std::cout << "\nS:\n";    print_(S, 1, N);
    std::cout << "\nVT:\n";   print_(VT, N, N);
    std::cout << "\n\n";

    matrix::multD(S, VT, W, N, M);
    matrix::mult(U, W, A_d, 1, 1, false, false, M, N, N);
    
    print_(A_d, M, N);
}

int main(){
    int N = 20;
    size_t size = N * sizeof(float);
    float T = 1e-9;
    float* voltage = (float*)malloc(size);
    float* d_voltage;
    gpuErrchk(hipMalloc(&d_voltage, size));

    // voltage[0] = 0;
    // for (int i = 1; i < N; i++){
    //     voltage[i] = voltage[i-1] * 0.99998000019999866667;
    //     if(rand()%10000 == 0)
    //         voltage[i] += 1;
    // }

    for (int i = 0; i < N; i++){
        voltage[i] = i;
    }

    voltage[0] = 100;
    for (int i = 1; i < N; i++){
        voltage[i] = voltage[i-1] * 0.99998000019999866667;
    }

    //Moving data to GPU
    hipMemcpy(d_voltage, voltage, size, hipMemcpyHostToDevice);

    //Fitting V = ae^(kt) + be^(qt) + c
    //param: {a, b, c, k, q}
    float param[5] = {
        1,
        -1,
        1,
        -1 * 1e-3,
        -10 * 1e-3
    };

    fit::FitGaussNewton(d_voltage, param, fit::doubleExp, N, 5);
    //TestMult();
    //TestSvd();
    hipFree(d_voltage);
    free(voltage);
    
}

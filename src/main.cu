#include "hip/hip_runtime.h"

#include <cstdlib>
#include <iostream>   
#include <iomanip>
#include <cmath>
#include <unistd.h>

#include "vector_wrappers.h"
#include "matrix_wrappers.h"
#include "cuda_tools.h"
#include "fitting.h"


//References:
//https://devblogs.nvidia.com/using-shared-memory-cuda-cc/
//https://docs.nvidia.com/cuda/cublas/index.html
//https://docs.nvidia.com/cuda/cusolver/index.html

namespace fit{

    // Applies lambda function to a vector
    template<typename f>
    __device__ float apply__(float A, float B, f lambda){
        return lambda(A, B);
    }
    template<typename f>
    __global__ void apply_(float* A, float B, float* C, f lambda, int N){
        int i = blockDim.x * blockIdx.x + threadIdx.x;
        if (i < N)
            C[i] = apply__(A[i], B, lambda);
    }
    template<typename f>
    __host__ void apply(float* A, float B, float* C, f lambda, int N){
        int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
        apply_<<<blocksPerGrid, threadsPerBlock>>>(A, B, C, lambda, N);
        gpuErrchk(hipDeviceSynchronize());
    }
    __host__ void invertS(float* S, float* invS, int N){
        auto lambda = []__device__(float a, float b){return (a > b) ? (1./a) : 0;};
        float eps = 1e-25 * vector::dot(S, S, N);
        //std::cout << eps << ":\n";
        //print_(S, 1, N);
        apply(S, eps, S, lambda, N);
        //print_(S, 1, N);
    }

    __host__ void pInvSVD(float* A, float* AInv, int M, int N){

        int P = M;
        if (M > N) P = N;

        float* S;       gpuErrchk(hipMalloc(&S, M * sizeof(float)));
        float* U;       gpuErrchk(hipMalloc(&U, M * M * sizeof(float)));
        float* VT;      gpuErrchk(hipMalloc(&VT, N * N * sizeof(float)));
        float* W;       gpuErrchk(hipMalloc(&W, M * N * sizeof(float)));
        gpuErrchk(hipMemset(W, 0, M * N * sizeof(float)));
        gpuErrchk(hipMemset(S, 0, M * sizeof(float)));
        svd(A, U, S, VT, M, N);

        matrix::multD(S, VT, W, N, M);
        matrix::mult(U, W, A, M, N, N);
        invertS(S, S, P);

        matrix::multD(S, VT, W, N, M);
        matrix::mult(U, W, A, M, N, N);
        matrix::transpose(A, AInv, M, N);

        if (S)  hipFree(S);
        if (U)  hipFree(U);
        if (VT) hipFree(VT);
        if (W)  hipFree(W);

    }

    __host__ void pInv(float* X, float* Inv, int M, int N){
        float* XT;  gpuErrchk(hipMalloc(&XT, M * N * sizeof(float)));
        float* XTX;  gpuErrchk(hipMalloc(&XTX, M * N * sizeof(float)));
        matrix::transpose(X, XT, M, N);
        matrix::mult(XT, X, XTX, N, M, N);
        matrix::inverse(XTX, XTX, N);
        matrix::mult(XTX, XT, Inv, N, N, M);
        if(XT)  hipFree(XT);
        if(XTX) hipFree(XTX);
    }

    //Gauss-Newton method
    __host__ void gaussNewton(float* A, float* param, void f(float* A, float* param, int N), int N, int K){

        float eps = 0.1;

        //Finding inverse of Jacobian
        float* J;       gpuErrchk(hipMalloc(&J, N * K * sizeof(float)));
        float* F;       gpuErrchk(hipMalloc(&F, N * sizeof(float)));
        float* P;       gpuErrchk(hipMalloc(&P, K * sizeof(float)));
        float* P_h =    (float*)malloc(K *sizeof(float));
        float error;
        int count = 0;
        do{
            count++;
            //J^-1
            jacobian_v2(J, f, param, N, K);
            pInvSVD(J, J, N, K);
            
            //(y - f(param))
            f(F, param, N);
            vector::sub(A, F, F, N);
            
            //J^-1(y - f(param))
            matrix::mult(J, F, P, K, N, 1);

            //Update param
            hipMemcpy(P_h, P, K * sizeof(float), hipMemcpyDeviceToHost);
            for (int i = 0; i < K; i++)
                param[i] += eps * P_h[i];
            
            //Calculate total squared error
            vector::mult(F, F, F, N);
            error = vector::sum(F, N)/N;

            //Print errors
            //print(param, 1, K);
            //std::cout << "Error:" << error << '\n';

        }while(error > 0.01);
        
        std::cout << "Converged in " << count << " iterations\n";
        if (J)  hipFree(J);
        if (F)  hipFree(F);
        if (P)  hipFree(P);
        if (P_h) free(P_h);
    }

    __host__ void robustLinear(float* Y, float* X, float* a, int N, int K){
        float* XT;  gpuErrchk(hipMalloc(&XT, N * K * sizeof(float)));
        float* U;   gpuErrchk(hipMalloc(&U, N * N * sizeof(float)));
        float* S;   gpuErrchk(hipMalloc(&S, K * sizeof(float)));
        float* VT;  gpuErrchk(hipMalloc(&VT, K * K * sizeof(float)));
        
        svd(X, U, S, VT, N, 2);
        invertS(S, S, 2);
        matrix::multD(S, VT, XT, 2, N);
        matrix::mult(U, XT, X, N, 2, 2);
        matrix::transpose(X, XT, N, 2);
        matrix::mult(XT, Y, a, 2, N, 1);

        if(XT)  hipFree(XT);
        if(U)   hipFree(U);
        if(S)   hipFree(S);
        if(VT)  hipFree(VT);
    }
    
    //Calculates (X^T X)^-1 X^T y
    //Y: Length N data series
    //X: N x K Parameter matrix
    //a: Length K fit coefficients
    __host__ void fastLinear(float* Y, float* X, float* a, int N, int K){
        float* XT;  gpuErrchk(hipMalloc(&XT, N * K * sizeof(float)));
        float* XTX;  gpuErrchk(hipMalloc(&XTX, N * K * sizeof(float)));
        matrix::transpose(X, XT, N, K);
        matrix::mult(XT, X, XTX, K, N, K);
        matrix::inverse(XTX, X, K);
        matrix::mult(X, XT, XTX, K, K, N);
        matrix::mult(XTX, Y, a, K, N, 1);
        if(XT)  hipFree(XT);
        if(XTX) hipFree(XTX);
    }
    
    //Fit Ae^(kt) + B
    //a = {A, B, k}
    __host__ void fastExpOffset(float* Y, float* param, int N){
        float* DY;  gpuErrchk(hipMalloc(&DY, (N-2) * sizeof(float)));
        float* X;   gpuErrchk(hipMalloc(&X, (N-2) * 2 * sizeof(float)));
        float* a;   gpuErrchk(hipMalloc(&a, 2 * sizeof(float)));
        float* fit; gpuErrchk(hipMalloc(&fit, N * sizeof(float)));

        //Compute Derivative
        vector::sub(Y, &Y[2], DY, N-2);
        
        //Linearize
        vector::log(DY, DY, N-2);
        //Fit exponential part
        lincoef(X, N-2);
        fastLinear(DY, X, a, N-2, 2);
        gpuErrchk(hipMemcpy(param, a, 2 * sizeof(float), hipMemcpyDeviceToHost));
        //Account for offset from numerical derivative
        param[1] -= param[0];

        //Rescaling constants for evaluating fit
        float temp = exp(param[1])/(-2*param[0]);
        param[1] = param[1] - log(abs(2 * param[0]));
        param[3] = 0;
        
        //Evaluating exponential fit
        flinear(fit, param, N);
        vector::exp(fit, fit, N);
        
        //Subtracting off fit to get constant offset
        vector::sub(Y, fit, fit, N);
        param[2] = vector::sum(fit, N)/N; 

        //Create output
        param[1] = temp;

        if(DY)  hipFree(DY);
        if(X)   hipFree(X);
        if(a)   hipFree(a);
        if(fit) hipFree(fit);
    }
    
}

void TestMult(){
    int M = 4;
    int N = 4;
    int K = 3;
    float A[16] = {3, 9, 2, 1, 1, 3, 5, 0, 3, 9, 2, 1, 1, 3, 5, 0};
    float B[12] = {2, 1, 2, 8, 9, 3, 4 ,1, 0, 5, 7, 5};
    float* A_d; gpuErrchk(hipMalloc(&A_d, M * N * sizeof(float)));
    float* B_d; gpuErrchk(hipMalloc(&B_d, N * K * sizeof(float)));
    float* C_d; gpuErrchk(hipMalloc(&C_d, M * K * sizeof(float)));
    hipMemcpy(A_d, &A, M * N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(B_d, &B, N * K * sizeof(float), hipMemcpyHostToDevice);
    matrix::mult(A_d, B_d, C_d, M, N, K);
    print_(C_d, M, K);
    std::cout << "\nTranspose: \n";
    matrix::transpose(C_d, C_d, M, K);
    print_(C_d, K, M);

    //Expected Output:

    // 21             43             31             
    // 63             129            93             
    // 53             46             64             
    // 4              13             7              
    
    // Transpose: 
    // 21             63             53             4              
    // 43             129            46             13             
    // 31             93             64             7   
}

void TestInv(){
    int N = 3;
    float A[9] = {1, 2, 4, 2, 3, 2, 3, 4, 1};
    float* A_d;     gpuErrchk(hipMalloc(&A_d, N * N * sizeof(float)));
    hipMemcpy(A_d, &A, N * N * sizeof(float), hipMemcpyHostToDevice);

    matrix::inverse(A_d, A_d, N);
    print_(A_d, N, N);
    
}

void TestSvd(){
    int M = 3;
    int N = 2;

    int P = M;
    if (M > N)
        P = N;

    float* result_h = (float*)malloc(M * N * sizeof(float)); 
    float* S;       gpuErrchk(hipMalloc(&S, P * sizeof(float)));
    float* U;       gpuErrchk(hipMalloc(&U, M * M * sizeof(float)));
    float* VT;      gpuErrchk(hipMalloc(&VT, N * N * sizeof(float)));
    float* W;       gpuErrchk(hipMalloc(&W, M * N * sizeof(float)));

    float A_h[20] = {
        2,0,0,
        0,-3,0};
    float* A_d;
    gpuErrchk(hipMalloc(&A_d, M * N * sizeof(float)));
    hipMemcpy(A_d, &A_h, M * N * sizeof(float), hipMemcpyHostToDevice);

    fit::svd(A_d, U, S, VT, M, N);
    
    std::cout << "\nU:\n";    print_(U, M, M);
    std::cout << "\nS:\n";    print_(S, 1, N);
    std::cout << "\nVT:\n";   print_(VT, N, N);
    std::cout << "\n\n";

    matrix::multD(S, VT, W, N, M);
    matrix::mult(U, W, A_d, M, N, N);
    
    print_(A_d, M, N);
}

void testLinFit(){
    int N = 5000;
    size_t size = N * sizeof(float);
    float* voltage = (float*)malloc(size);
    float* d_voltage;   gpuErrchk(hipMalloc(&d_voltage, size));
    float* X;           gpuErrchk(hipMalloc(&X, N * 2 * sizeof(float)));
    float* a;           gpuErrchk(hipMalloc(&a, 2 * sizeof(float)));
    float milliseconds = 0;
    hipEvent_t start, stop;
    gpuErrchk(hipEventCreate(&start));
    gpuErrchk(hipEventCreate(&stop));

    voltage[0] = 100;
    for (int i = 1; i < N; i++)
        voltage[i] = voltage[i-1] - 0.5;
    hipMemcpy(d_voltage, voltage, size, hipMemcpyHostToDevice);

    std::cout << "\n\nFitting linear (2 parameter linear fit).\nV = at + b\nExpected Parameters:\n-0.5, 100 \n";
    std::cout << "Result: \n";

    hipDeviceSynchronize();
    hipEventRecord(start);
    for (int i = 0; i < 100; i++){
        fit::lincoef(X, N);
        fit::robustLinear(d_voltage, X, a, N, 2);
    }
    hipDeviceSynchronize();
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    std::cout << milliseconds << "ms elapsed.\n";
    print_(a, 1, 2);
    
    hipDeviceSynchronize();
    hipEventRecord(start);
    for (int i = 0; i < 100; i++){
        fit::lincoef(X, N);
        fit::fastLinear(d_voltage, X, a, N, 2);
    }
    hipDeviceSynchronize();
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    std::cout << milliseconds << "ms elapsed.\n";
    print_(a, 1, 2);

    hipEventDestroy(start);
    hipEventDestroy(stop);
    if(d_voltage)   hipFree(d_voltage);
    if(X)           hipFree(X);
    if(a)           hipFree(a);
    if(voltage)     free(voltage);
}

void testExpFit(){
    int N = 5000;
    size_t size = N * sizeof(float);
    float* voltage = (float*)malloc(size);
    float* param = (float*)malloc(3 * sizeof(float));
    float* d_voltage;   gpuErrchk(hipMalloc(&d_voltage, size));
    float* X;           gpuErrchk(hipMalloc(&X, size * 2));
    float* a;           gpuErrchk(hipMalloc(&a, 2 * sizeof(float)));
    float milliseconds = 0;
    hipEvent_t start, stop;
    gpuErrchk(hipEventCreate(&start));
    gpuErrchk(hipEventCreate(&stop));
    
    voltage[0] = 100;
    for (int i = 1; i < N; i++)
        voltage[i] = voltage[i-1] * 0.999;
    for (int i = 1; i < N; i++)
        voltage[i] += .01;

    hipMemcpy(d_voltage, voltage, size, hipMemcpyHostToDevice);
    
    std::cout << "\n\nFitting Exponential (3 parameter linear fit).\nV = at + b\nExpected Parameters:\n-0.00010005, 100, 0.01\n";
    std::cout << "Result: \n";

    hipDeviceSynchronize();
    hipEventRecord(start);
    for (int i = 0; i < 100; i++)
        fit::fastExpOffset(d_voltage, param, N);
    hipDeviceSynchronize();
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    std::cout << milliseconds << "ms elapsed.\n";
    print(param, 1, 3);

    hipEventDestroy(start);
    hipEventDestroy(stop);
    if(d_voltage)   hipFree(d_voltage);
    if(X)           hipFree(X);
    if(a)           hipFree(a);
    if(voltage)     free(voltage);
    if(param)       free(param);
}

void testGaussNewtonFit(){
    int N = 5000;
    size_t size = N * sizeof(float);
    float T = 1e-9;
    float* voltage = (float*)malloc(size);
    float* d_voltage;   gpuErrchk(hipMalloc(&d_voltage, size));
    float* d_fit;       gpuErrchk(hipMalloc(&d_fit, size));
    float milliseconds = 0;
    int failed;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    voltage[0] = 100;
    for (int i = 1; i < N; i++)
        voltage[i] = voltage[i-1] * .999;
    for (int i = 1; i < N; i++){
        voltage[i] -= exp(-0.01 * i) + 0.01;
    }
    hipMemcpy(d_voltage, voltage, size, hipMemcpyHostToDevice);

    float param[5];
   
    srand (time(NULL));

    failed = 0;
    std::cout << "\n\n\nFitting double exponential (5 parameter non-linear fit).\nV = ae^(kt) + be^(qt) + c\n"; 
    hipDeviceSynchronize();
    hipEventRecord(start);
    for (int i = 0; i < 10; i++){
        param[0] = rand() % 10000; // From 1 to 1e4
        param[1] = (-rand() % 10000) * 1e-2; // From -1e-2 to -1e2
        param[2] = rand()%1001 - 500; // From -1e2 to 1e2
        param[3] = -rand()%10000 * 1e-5; // From 1e-5 to 0.1
        param[4] = -rand()%10000 * 1e-3; // From 1e-3 to 10
        fit::gaussNewton(d_voltage, param, fit::fdoubleExp, N, 5);
        for (int j = 0; j < 5; j++)
            if (isnan(param[j]) || isinf(param[j])){
                failed += 1;
                break;
            }
    }
    hipDeviceSynchronize();
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    std::cout << milliseconds << "ms elapsed." << "\n";
    std::cout << "Failed " << failed << " times\n";
    std::cout << "Expected parameters:\n 100, -1, -0.01, -0.0010005, -0.01\n";
    std::cout << "Result: \n";
    print(param, 1, 5);



    voltage[0] = 100;
    for (int i = 1; i < N; i++)
        voltage[i] = voltage[i-1] * .999;
    for (int i = 1; i < N; i++)
        voltage[i] -= 0.01;

    failed = 0;
    std::cout << "\n\n\nFitting exponential (3 parameter non-linear fit).\nV = ae^(kt) + c\n"; 
    hipDeviceSynchronize();
    hipEventRecord(start);
    for (int i = 0; i < 10; i++){
        param[0] = rand() % 10000 + 1; // From 1 to 10000
        param[1] = rand()%1001 - 500; // From -500 to 500
        param[2] = -rand()%10000 * 1e-5; // From 1e-5 to 0.1
        fit::gaussNewton(d_voltage, param, fit::fexp, N, 3);
        for (int j = 0; j < 3; j++)
            if (isnan(param[j]) || isinf(param[j])){
                failed += 1;
                break;
            }
    }
    hipDeviceSynchronize();
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    std::cout << milliseconds << "ms elapsed." << "\n";
    std::cout << "Failed " << failed << " times\n";
    std::cout << "Expected parameters:\n 100, -0.01, -0.0010005\n";
    std::cout << "Result: \n";
    print(param, 1, 3);



    voltage[0] = 100;
    for (int i = 1; i < N; i++)
        voltage[i] = voltage[i-1] - 0.5;
    hipMemcpy(d_voltage, voltage, size, hipMemcpyHostToDevice);

    std::cout << "\n\n\nFitting linear (2 parameter non-linear fit).\nV = ax + b\n"; 
    hipDeviceSynchronize();
    hipEventRecord(start);
    for (int i = 0; i < 10; i++){
        param[0] = -0.5 * (rand() % 10000) * 1e-2;
        param[1] = rand() % 10000 - 5000;
        fit::gaussNewton(d_voltage, param, fit::flinear, N, 2);
        for (int j = 0; j < 2; j++)
            if (isnan(param[j]) || isinf(param[j])){
                failed += 1;
                break;
            }
    }
    hipDeviceSynchronize();
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    std::cout << milliseconds << "ms elapsed." << "\n";
    std::cout << "Failed " << failed << " times\n";
    std::cout << "Expected parameters:\n -0.5, 100,\n";
    std::cout << "Result: \n";
    print(param, 1, 2);

    hipEventDestroy(start);
    hipEventDestroy(stop);
    hipFree(d_voltage);
    free(voltage);
}

int main(){

    // testLinFit();
    // testExpFit();
    testGaussNewtonFit();
}

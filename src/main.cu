
#include <cstdlib>
#include <chrono>
#include <iostream>   
#include <iomanip>
#include <cmath>

#include "vector_wrappers.h"
#include "matrix_wrappers.h"
#include "fitting.h"
#include "cuda_tools.h"



//References:
//https://devblogs.nvidia.com/using-shared-memory-cuda-cc/
//https://docs.nvidia.com/cuda/cublas/index.html
//https://docs.nvidia.com/cuda/cusolver/index.html


namespace fit{

    __host__ void MPInverse(float* A, int M, int N){

        //Cusolver SVD requires M >= N

        float* S; gpuErrchk(hipMalloc(&S, N * N * sizeof(float)));
        float* U; gpuErrchk(hipMalloc(&U, M * M * sizeof(float)));
        float* VT; gpuErrchk(hipMalloc(&VT, N * N * sizeof(float)));

        svd(A, M, N, S, U, VT);
        
    }

    //Gauss-Newton method
    __host__ void FitGaussNewton(float* A, float* param, float* f(float* A, float* param, int N), int N, int K){

        int P = N;
        if (N > K) P = K;

        //Finding inverse of Jacobian
        float* J;       gpuErrchk(hipMalloc(&J, N * K * sizeof(float)));
        float* S;       gpuErrchk(hipMalloc(&S, P * P * sizeof(float)));
        float* U;      gpuErrchk(hipMalloc(&U, N * N * sizeof(float)));
        float* VT;     gpuErrchk(hipMalloc(&VT, K * K * sizeof(float)));
        
 
        
        jacobian(J, f, param, N, K);
        svd(J, N, K, S, U, VT);


        // hipblasSgeam(handle,
        //     HIPBLAS_OP_T, HIPBLAS_OP_N,
        //     int m, int n,
        //     const float           *alpha,
        //     const float           *A, int lda,
        //     const float           *beta,
        //     const float           *B, int ldb,
        //     float           *C, int ldc);

        hipFree(S);
        hipFree(U);
        hipFree(VT);
    }

}

__host__ void print_(const float* A_d, int M, int N){
    float* A_h = (float*)malloc(M * N * sizeof(float));
    gpuErrchk(hipMemcpy(A_h, A_d, M * N * sizeof(float), hipMemcpyDeviceToHost));
    for (int i = 0; i < M; i++){
        for (int j = 0; j < N; j++){
            std::cout << std::left << std::setw(15) << A_h[i + j * M];
        }
        std::cout << '\n';
    }
}
__host__ void print(const float* A_h, int M, int N){
    for (int i = 0; i < M; i++){
        for (int j = 0; j < N; j++){
            std::cout << std::left << std::setw(15) << A_h[i + j * M];
        }
        std::cout << '\n';
    }
}

void TestMult(){
    int M = 2;
    int N = 4;
    int K = 3;
    float A[8] = {3, 9, 2, 1, 1, 3, 5, 0};
    float B[12] = {2, 1, 2, 8, 9, 3, 4 ,1, 0, 5, 7, 5};
    float* A_d; gpuErrchk(hipMalloc(&A_d, M * N * sizeof(float)));
    float* B_d; gpuErrchk(hipMalloc(&B_d, N * K * sizeof(float)));
    float* C_d; gpuErrchk(hipMalloc(&C_d, M * K * sizeof(float)));
    hipMemcpy(A_d, &A, M * N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(B_d, &B, N * K * sizeof(float), hipMemcpyHostToDevice);
    matrix::MultMat(A_d, B_d, C_d, 1, 1, M, N, K);
    

    print_(C_d, M, K);
}

void TestSvd(){
    int M = 3;
    int N = 2;

    int P = M;
    if (M > N)
        P = N;

    float* result_h = (float*)malloc(M * N * sizeof(float)); 
    float* S;       gpuErrchk(hipMalloc(&S, P * sizeof(float)));
    float* U;       gpuErrchk(hipMalloc(&U, M * M * sizeof(float)));
    float* VT;      gpuErrchk(hipMalloc(&VT, N * N * sizeof(float)));
    float* W;       gpuErrchk(hipMalloc(&W, M * N * sizeof(float)));

    float A_h[20] = {
        2,0,0,
        0,-3,0};
    float* A_d;
    gpuErrchk(hipMalloc(&A_d, M * N * sizeof(float)));
    hipMemcpy(A_d, &A_h, M * N * sizeof(float), hipMemcpyHostToDevice);

    fit::svd(A_d, M, N, S, U, VT);
    
    std::cout << "\nU:\n";    print_(U, M, M);
    std::cout << "\nS:\n";    print_(S, 1, N);
    std::cout << "\nVT:\n";   print_(VT, N, N);
    std::cout << "\n\n";

    matrix::MultMatD(S, VT, W, N, N);
    matrix::MultMat(U, W, A_d, 1, 1, M, N, N);
    
    print_(A_d, M, N);
}

int main(){
    int N = 1000;
    size_t size = N * sizeof(float);
    float T = 1e-9;
    float* voltage = (float*)malloc(size);
    float* d_voltage;
    gpuErrchk(hipMalloc(&d_voltage, size));

    // voltage[0] = 0;
    // for (int i = 1; i < N; i++){
    //     voltage[i] = voltage[i-1] * 0.99998000019999866667;
    //     if(rand()%10000 == 0)
    //         voltage[i] += 1;
    // }

    for (int i = 0; i < N; i++){
        voltage[i] = i;
    }

    voltage[0] = 100;
    for (int i = 1; i < N; i++){
        voltage[i] = voltage[i-1] * 0.99998000019999866667;
    }

    //Moving data to GPU
    hipMemcpy(d_voltage, voltage, size, hipMemcpyHostToDevice);

    //Fitting V = ae^(kt) + be^(qt) + c
    //param: {a, b, c, k, q}
    float param[5] = {
        1,
        -1,
        1,
        -1 * 1e-3,
        -10 * 1e-3
    };



    TestSvd();



    hipFree(d_voltage);
    free(voltage);
    
}

#include "hip/hip_runtime.h"

#include <cstdlib>
#include <iostream>   
#include <iomanip>
#include <cmath>

#include "vector_wrappers.h"
#include "matrix_wrappers.h"
#include "cuda_tools.h"
#include "fitting.h"


//References:
//https://devblogs.nvidia.com/using-shared-memory-cuda-cc/
//https://docs.nvidia.com/cuda/cublas/index.html
//https://docs.nvidia.com/cuda/cusolver/index.html

namespace fit{

    // Applies lambda function to a vector
    template<typename f>
    __device__ float apply__(float A, float B, f lambda){
        return lambda(A, B);
    }
    template<typename f>
    __global__ void apply_(float* A, float B, float* C, f lambda, int N){
        int i = blockDim.x * blockIdx.x + threadIdx.x;
        if (i < N)
            C[i] = apply__(A[i], B, lambda);
    }
    template<typename f>
    __host__ void apply(float* A, float B, float* C, f lambda, int N){
        int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
        apply_<<<blocksPerGrid, threadsPerBlock>>>(A, B, C, lambda, N);
        gpuErrchk(hipDeviceSynchronize());
    }
    __host__ void invertS(float* S, float* invS, int N){
        auto lambda = []__device__(float a, float b){return (a > b) ? (1./a) : 0;};
        float eps = 1e-25 * vector::dot(S, S, N);
        //std::cout << eps << ":\n";
        //print_(S, 1, N);
        apply(S, eps, S, lambda, N);
        //print_(S, 1, N);
    }

    __host__ void pInv(float* A, int M, int N){

        int P = M;
        if (M > N) P = N;

        float* S;       gpuErrchk(hipMalloc(&S, M * sizeof(float)));
        float* U;       gpuErrchk(hipMalloc(&U, M * M * sizeof(float)));
        float* VT;      gpuErrchk(hipMalloc(&VT, N * N * sizeof(float)));
        float* W;       gpuErrchk(hipMalloc(&W, M * N * sizeof(float)));
        gpuErrchk(hipMemset(W, 0, M * N * sizeof(float)));
        gpuErrchk(hipMemset(S, 0, M * sizeof(float)));
        svd(A, U, S, VT, M, N);

        matrix::multD(S, VT, W, N, M);
        matrix::mult(U, W, A, 1, 1, false, false, M, N, N);
        invertS(S, S, P);


        matrix::multD(S, VT, W, N, M);
        matrix::mult(U, W, A, 1, 1, false, false, M, N, N);
        matrix::transpose(A, A, M, N);

        if (S)  hipFree(S);
        if (U)  hipFree(U);
        if (VT) hipFree(VT);
        if (W)  hipFree(W);

    }

    //Gauss-Newton method
    __host__ void gaussNewton(float* A, float* param, void f(float* A, float* param, int N), int N, int K){

        float eps = 0.05;

        //Finding inverse of Jacobian
        float* J;       gpuErrchk(hipMalloc(&J, N * K * sizeof(float)));
        float* F;       gpuErrchk(hipMalloc(&F, N * sizeof(float)));
        float* P;       gpuErrchk(hipMalloc(&P, K * sizeof(float)));
        float* P_h =    (float*)malloc(K *sizeof(float));
        float error;
        int count = 0;
        do{
            count++;
            //J^-1
            jacobian_v2(J, f, param, N, K);
            pInv(J, N, K);
            
            //(y - f(param))
            doubleExp(F, param, N);
            vector::sub(A, F, F, N);
            
            //J^-1(y - f(param))
            matrix::mult(J, F, P, 1, 1, false, false, K, N, 1);

            //Update param
            hipMemcpy(P_h, P, K * sizeof(float), hipMemcpyDeviceToHost);
            for (int i = 0; i < K; i++)
                param[i] += eps * P_h[i];
            
            //Print errors
            // doubleExp(F, param, N);
            // vector::sub(A, F, F, N);
            // std::cout << "Error:" << vector::sum(F, N) << '\n';

            error = vector::sum(F, N)/N;

        }while(error > 0.001);
        
        std::cout << "Converged in " << count << " iterations\n";
        if (J)  hipFree(J);
        if (F)  hipFree(F);
        if (P)  hipFree(P);
        if (P_h) free(P_h);
    }

    __host__ void robust(float* Y, float* a, int N){
        float* X;   gpuErrchk(hipMalloc(&X, N * 2 * sizeof(float)));
        float* XT;  gpuErrchk(hipMalloc(&XT, N * 2 * sizeof(float)));
        float* U;   gpuErrchk(hipMalloc(&U, N * N * sizeof(float)));
        float* S;   gpuErrchk(hipMalloc(&S, 2 * sizeof(float)));
        float* VT;  gpuErrchk(hipMalloc(&VT, 2 * 2 * sizeof(float)));
        lincoef(X, N);
        svd(X, U, S, VT, N, 2);
        invertS(S, S, 2);
        matrix::multD(S, VT, XT, 2, N);
        matrix::mult(U, XT, X, 1, 1, false, false, N, 2, 2);
        matrix::transpose(X, XT, N, 2);
        matrix::mult(XT, Y, a, 1, 1, false, false, 2, N, 1);
    }

}



void TestMult(){
    int M = 4;
    int N = 4;
    int K = 3;
    float A[16] = {3, 9, 2, 1, 1, 3, 5, 0, 3, 9, 2, 1, 1, 3, 5, 0};
    float B[12] = {2, 1, 2, 8, 9, 3, 4 ,1, 0, 5, 7, 5};
    float* A_d; gpuErrchk(hipMalloc(&A_d, M * N * sizeof(float)));
    float* B_d; gpuErrchk(hipMalloc(&B_d, N * K * sizeof(float)));
    float* C_d; gpuErrchk(hipMalloc(&C_d, M * K * sizeof(float)));
    hipMemcpy(A_d, &A, M * N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(B_d, &B, N * K * sizeof(float), hipMemcpyHostToDevice);
    matrix::mult(A_d, B_d, C_d, 1, 1, false, false, M, N, K);
    print_(C_d, M, K);
    std::cout << "\nTranspose: \n";
    matrix::transpose(C_d, C_d, M, K);
    print_(C_d, K, M);

    //Expected Output:

    // 21             43             31             
    // 63             129            93             
    // 53             46             64             
    // 4              13             7              
    
    // Transpose: 
    // 21             63             53             4              
    // 43             129            46             13             
    // 31             93             64             7   
}

void TestInv(){
    int N = 3;
    float A[9] = {1, 2, 4, 2, 3, 2, 3, 4, 1};
    float* A_d;     gpuErrchk(hipMalloc(&A_d, N * N * sizeof(float)));
    hipMemcpy(A_d, &A, N * N * sizeof(float), hipMemcpyHostToDevice);

    matrix::inverse(A_d, A_d, N);
    print_(A_d, N, N);
    
}

void TestSvd(){
    int M = 3;
    int N = 2;

    int P = M;
    if (M > N)
        P = N;

    float* result_h = (float*)malloc(M * N * sizeof(float)); 
    float* S;       gpuErrchk(hipMalloc(&S, P * sizeof(float)));
    float* U;       gpuErrchk(hipMalloc(&U, M * M * sizeof(float)));
    float* VT;      gpuErrchk(hipMalloc(&VT, N * N * sizeof(float)));
    float* W;       gpuErrchk(hipMalloc(&W, M * N * sizeof(float)));

    float A_h[20] = {
        2,0,0,
        0,-3,0};
    float* A_d;
    gpuErrchk(hipMalloc(&A_d, M * N * sizeof(float)));
    hipMemcpy(A_d, &A_h, M * N * sizeof(float), hipMemcpyHostToDevice);

    fit::svd(A_d, U, S, VT, M, N);
    
    std::cout << "\nU:\n";    print_(U, M, M);
    std::cout << "\nS:\n";    print_(S, 1, N);
    std::cout << "\nVT:\n";   print_(VT, N, N);
    std::cout << "\n\n";

    matrix::multD(S, VT, W, N, M);
    matrix::mult(U, W, A_d, 1, 1, false, false, M, N, N);
    
    print_(A_d, M, N);
}

void TestLinFit(){
    int N = 100;
    size_t size = N * sizeof(float);
    float* voltage = (float*)malloc(size);
    float* d_voltage;   gpuErrchk(hipMalloc(&d_voltage, size));
    float* a;           gpuErrchk(hipMalloc(&a, 2 * sizeof(float)));
    voltage[0] = 100;
    for (int i = 1; i < N; i++)
        voltage[i] = voltage[i-1] - 0.5;
    hipMemcpy(d_voltage, voltage, size, hipMemcpyHostToDevice);
    float milliseconds = 0;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    fit::robust(d_voltage, a, N);
    hipEventRecord(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    std::cout << milliseconds << "ms elapsed.\n";
    print_(a, 1, 2);
}

int main(){
    int N = 5000;
    size_t size = N * sizeof(float);
    float T = 1e-9;
    float* voltage = (float*)malloc(size);
    float* d_voltage;   gpuErrchk(hipMalloc(&d_voltage, size));
    float* d_fit;       gpuErrchk(hipMalloc(&d_fit, size));
    float milliseconds = 0;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // voltage[0] = 0;
    // for (int i = 1; i < N; i++){
    //     voltage[i] = voltage[i-1] * 0.99998000019999866667;
    //     if(rand()%10000 == 0)
    //         voltage[i] += 1;
    // }

    voltage[0] = 100;
    
    for (int i = 1; i < N; i++){
        voltage[i] = voltage[i-1] * .99998;
    }

    hipMemcpy(d_voltage, voltage, size, hipMemcpyHostToDevice);

    //Fitting V = ae^(kt) + be^(qt) + c
    //param: {a, b, c, k, q}
    std::cout << "\n\n\nFitting double exponential (5 parameter non-linear fit).\nV = ae^(kt) + be^(qt) + c\n"; 
    float param[5] = {
        1,
        -1,
        1,
        -1 * 1e-3,
        -10 * 1e-3
    };
    
    hipEventRecord(start);
    fit::gaussNewton(d_voltage, param, fit::doubleExp, N, 5);
    hipEventRecord(stop);
    fit::doubleExp(d_fit, param, N);
    hipEventElapsedTime(&milliseconds, start, stop);
    std::cout << milliseconds << "ms elapsed." << "\n";

    //print_(d_fit, N, 1);
    //print_(d_voltage, N, 1);
    std::cout << "Expected parameters:\n 100, 0, 0, -0.00002, <undefined>\n";
    std::cout << "Result: \n";
    print(param, 1, 5);
    
    //TestMult();
    //TestSvd();
    //TestInv();
    std::cout << "\n\nFitting linear (2 parameter linear fit).\nV = at + b\nExpected Parameters:\n-0.5, 100 \n";
    std::cout << "Result: \n";
    TestLinFit();
    hipFree(d_voltage);
    free(voltage);
}

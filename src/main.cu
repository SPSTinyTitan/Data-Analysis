#include "hip/hip_runtime.h"

#include <cstdlib>
#include <chrono>
#include <iostream>   
#include <iomanip>
#include "vector.h"
#include <cmath>

//References:
//https://devblogs.nvidia.com/using-shared-memory-cuda-cc/
//https://docs.nvidia.com/cuda/cublas/index.html
//https://docs.nvidia.com/cuda/cusolver/index.html

#include <cstdio>
#include <hip/hip_runtime.h>
#include <assert.h>
#include <hipsolver.h>
#include <hipblas.h>

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true){
    if (code != hipSuccess) {
        fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) 
            exit(code);
    }
}

namespace vector {


//Return: ae^(kt) + be^(qt) + c
__global__ void DoubleExp(float* A, float a, float b, float c, float k, float q, int N){
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < N)
        A[i] = a * expf(k * i) + b * expf(q * i) + c;
}
__host__ float* DoubleExp(float* param, int N){
    float* A;
    gpuErrchk(hipMalloc(&A, N*sizeof(float)));
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
    DoubleExp<<<blocksPerGrid, threadsPerBlock>>>(A, param[0], param[1], param[2], param[3], param[4], N);
    return A;
}

//Copy vector. A = B; 
__global__ void CopyVecf(float* A, float* B, int N){
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < N)
        A[i] = B[i];
}
__host__ void CopyVecfh(float* A, float* B, int N){
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
    CopyVecf<<<blocksPerGrid, threadsPerBlock>>>(A, B, N);
}

//Calculates the Jacobian of f(t) for K parameters and N data points
//Returns N x K matrix in column major order

__host__ float* Jacobian(float* f(float* param, int N), float* param, int N, int K){
    const float eps = pow(2,-17);
    float* J;
    gpuErrchk(hipMalloc(&J, N * K * sizeof(float)));

    float* prev = f(param, N);
    float* next;

    float* debug = (float*)malloc(N * K * sizeof(float));


    for (int i = 0; i < K; i++){
        param[i] += eps;
        next = f(param, N);
        SubVecfh(next, prev, (float*)&J[i * N], N);
        param[i] -= eps;
        hipFree(next);
    }
    
    hipFree(prev);
    return ScaleVecf(J, float(1./eps), N * K);
}

//Same as Jacboian() but uses adaptive values of epsilon scaled with the value of param.
//This can achieve better precisions across a wider range of values.
//TODO: Consider logarithms then addition instead of multiplications. Reduces arithmetic error.
__host__ float* Jacobian2(float* f(float* param, int N), float* param, int N, int K){
    const float eps = pow(2,-10);
    float* J;
    gpuErrchk(hipMalloc(&J, N * K * sizeof(float)));

    float* prev = f(param, N);
    float* next;
    float temp, dB, scale;

    for (int i = 0; i < K; i++){
        temp = param[i];
        dB = param[i] * eps;
        if (dB < 1e-31 && dB > -1e-31)
            dB = 1e-30;
        //dB^-1. Recalculation for precision. 
        scale = (float) (1./((double)param[i] * eps));
        param[i] += dB;
        next = f(param, N);
        SubVecfh(next, prev, (float*)&J[i * N], N);
        ScaleVecfh((float*)&J[i * N], scale, (float*)&J[i * N], N);
        param[i] = temp;
        hipFree(next);
    }
    
    hipFree(prev);
    return J;
}
//C = (alpha * A) x (beta * B)
//A - M x N
//B - N x K
//C - M x K
__host__ void MultMat(float* A, float* B, float* C, float alpha, float beta, int M, int N, int K){

    //Initializing CuBlas
    hipblasHandle_t cublasH = NULL;
    hipblasStatus_t cublas_status = HIPBLAS_STATUS_SUCCESS;
    cublas_status = hipblasCreate(&cublasH);
    assert(HIPBLAS_STATUS_SUCCESS == cublas_status);

    //Zeroing results
    hipMemset(C, 0, M * K * sizeof(float));

    //Multiplying
    cublas_status = hipblasSgemm(
        cublasH,
        HIPBLAS_OP_N, HIPBLAS_OP_N,
        M, K, N,
        &alpha,
        A, M,
        B, N,
        &beta,
        C, M
    );
    assert(HIPBLAS_STATUS_SUCCESS == cublas_status);
}

//Calculates diagonal matrix multiplication
//C = A x B
//A - 1 x N diagonal (representing MxN)
//B - N x K
__host__ void MultMatD(float* A, float* B, float* C, int M, int N){
     
    //Initializing CuBlas
    hipblasHandle_t cublasH = NULL;
    hipblasStatus_t cublas_status = HIPBLAS_STATUS_SUCCESS;
    cublas_status = hipblasCreate(&cublasH);
    assert(HIPBLAS_STATUS_SUCCESS == cublas_status);

    //Zeroing results
    hipMemset(C, 0, M * N * sizeof(float));

    //Multiplying
    cublas_status = hipblasSdgmm(
        cublasH, HIPBLAS_SIDE_LEFT,
        M, N,
        B, M,
        A, 1,
        C, M
    );
    assert(HIPBLAS_STATUS_SUCCESS == cublas_status);
}

__host__ void SVD(float* A, int M, int N, float* S, float* U, float* VT){ 
    
    // float* debug = (float*)malloc(M * N * sizeof(float));
    // hipMemcpy(debug, A, M * N * sizeof(float), hipMemcpyDeviceToHost);
    // for (int i = 0; i < M; i++){
    //     for (int j = 0; j < N; j++){
    //         std::cout << std::left << std::setw(15) << debug[i + j * M];
    //     }
    //     std::cout << '\n';
    // }
    
    hipsolverHandle_t handle;
    hipsolverStatus_t stat = hipsolverDnCreate(&handle);
    if (stat != HIPSOLVER_STATUS_SUCCESS) {
        printf ("CUSOLVER initialization failed\n");
        exit(EXIT_FAILURE);
    }

    int* devInfo; gpuErrchk(hipMalloc(&devInfo, sizeof(int))); 
    int lwork;

    stat = hipsolverDnSgesvd_bufferSize(
        handle,
        M,
        N,
        &lwork);
    if (stat != HIPSOLVER_STATUS_SUCCESS) {
        printf ("SVD buffer failed\n");
        exit(EXIT_FAILURE);
    }

    float* work; gpuErrchk(hipMalloc(&work, lwork * sizeof(float)));

    stat = hipsolverDnSgesvd(
        handle,
        'A',
        'A',
        M,
        N,
        A,
        M,
        S,
        U,
        M,
        VT,
        N,
        work,
        lwork,
        NULL,
        devInfo);

    if (stat != HIPSOLVER_STATUS_SUCCESS) {
        printf ("SVD Solve failed\n");
        int dev = 0;
        hipMemcpy(&dev, devInfo, sizeof(int), hipMemcpyDeviceToHost);
        printf ("Code: %d, Devinfo: %d \n", (int)stat, dev);
        printf ("M:%d N:%d lda:%d ldu:%d ldvt:%d", M, N, M, M, N);
        exit(EXIT_FAILURE);
    }

    hipFree(work);
    hipFree(devInfo);
}

__host__ void MPInverse(float* A, int M, int N){

    //Cusolver SVD requires M >= N

    float* S; gpuErrchk(hipMalloc(&S, N * N * sizeof(float)));
    float* U; gpuErrchk(hipMalloc(&U, M * M * sizeof(float)));
    float* VT; gpuErrchk(hipMalloc(&VT, N * N * sizeof(float)));

    SVD(A, M, N, S, U, VT);
    
}

//Gauss-Newton method
__host__ void FitGaussNewton(float* A, float* param, float* f(float* param, int N), int N, int K){

    //Finding inverse Jacobian
    float* J = Jacobian(f, param, N, K);
    float* S, *U, *VT;
    int P = N;
    if (N > K)
        P = K;

    gpuErrchk(hipMalloc(&S, P * P * sizeof(float)));
    gpuErrchk(hipMalloc(&U, N * N * sizeof(float)));
    gpuErrchk(hipMalloc(&VT, K * K * sizeof(float)));
    SVD(J, N, K, S, U, VT);


    // hipblasSgeam(handle,
    //     HIPBLAS_OP_T, HIPBLAS_OP_N,
    //     int m, int n,
    //     const float           *alpha,
    //     const float           *A, int lda,
    //     const float           *beta,
    //     const float           *B, int ldb,
    //     float           *C, int ldc);

    hipFree(S);
    hipFree(U);
    hipFree(VT);
}

}

__host__ void Print(const float* A, int M, int N){
    for (int i = 0; i < M; i++){
        for (int j = 0; j < N; j++){
            std::cout << std::left << std::setw(15) << A[i + j * M];
        }
        std::cout << '\n';
    }
}

void TestMult(){
    int M = 2;
    int N = 4;
    int K = 3;
    float A[8] = {3, 9, 2, 1, 1, 3, 5, 0};
    float B[12] = {2, 1, 2, 8, 9, 3, 4 ,1, 0, 5, 7, 5};
    float* C = (float*)malloc(M * K * sizeof(float));
    float* A_d; gpuErrchk(hipMalloc(&A_d, M * N * sizeof(float)));
    float* B_d; gpuErrchk(hipMalloc(&B_d, N * K * sizeof(float)));
    float* C_d; gpuErrchk(hipMalloc(&C_d, M * K * sizeof(float)));
    hipMemcpy(A_d, &A, M * N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(B_d, &B, N * K * sizeof(float), hipMemcpyHostToDevice);
    vector::MultMat(A_d, B_d, C_d, 1, 1, M, N, K);
    
    hipMemcpy(C, C_d, M * K * sizeof(float), hipMemcpyDeviceToHost);
    

    Print(C, M, K);
}

void TestSVD(){
    int M = 3;
    int N = 2;

    int P = M;
    if (M > N)
        P = N;

    float* Sh =     (float*)malloc(P * sizeof(float));
    float* Uh =     (float*)malloc(M * M * sizeof(float));
    float* VTh =    (float*)malloc(N * N * sizeof(float));
    float* result_h=(float*)malloc(M * N * sizeof(float)); 
    float* S;       gpuErrchk(hipMalloc(&S, P * sizeof(float)));
    float* U;       gpuErrchk(hipMalloc(&U, M * M * sizeof(float)));
    float* VT;      gpuErrchk(hipMalloc(&VT, N * N * sizeof(float)));
    float* W;       gpuErrchk(hipMalloc(&W, M * N * sizeof(float)));

    float A_h[20] = {
        2,0,0,
        0,-3,0};
    float* A_d;
    gpuErrchk(hipMalloc(&A_d, M * N * sizeof(float)));
    hipMemcpy(A_d, &A_h, M * N * sizeof(float), hipMemcpyHostToDevice);

    vector::SVD(A_d, M, N, S, U, VT);
    
    hipMemcpy(Sh, S, P * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(Uh, U, M * M * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(VTh, VT, N * N * sizeof(float), hipMemcpyDeviceToHost);
    
    std::cout << "\nU:\n";    Print(Uh, M, M);
    std::cout << "\nS:\n";    Print(Sh, 1, N);
    std::cout << "\nVT:\n";   Print(VTh, N, N);
    std::cout << "\n\n";

    vector::MultMatD(S, VT, W, N, N);
    vector::MultMat(U, W, A_d, 1, 1, M, N, N);
    
    hipMemcpy(result_h, A_d, M * N * sizeof(float), hipMemcpyDeviceToHost);
    Print(result_h, M, N);
}

int main(){
    int N = 1000;
    size_t size = N * sizeof(float);
    float T = 1e-9;
    float* voltage = (float*)malloc(size);
    float* d_voltage;
    gpuErrchk(hipMalloc(&d_voltage, size));

    // voltage[0] = 0;
    // for (int i = 1; i < N; i++){
    //     voltage[i] = voltage[i-1] * 0.99998000019999866667;
    //     if(rand()%10000 == 0)
    //         voltage[i] += 1;
    // }

    for (int i = 0; i < N; i++){
        voltage[i] = i;
    }

    voltage[0] = 100;
    for (int i = 1; i < N; i++){
        voltage[i] = voltage[i-1] * 0.99998000019999866667;
    }

    //Moving data to GPU
    hipMemcpy(d_voltage, voltage, size, hipMemcpyHostToDevice);

    //Fitting V = ae^(kt) + be^(qt) + c
    //param: {a, b, c, k, q}
    float param[5] = {
        1,
        -1,
        1,
        -1 * 1e-3,
        -10 * 1e-3
    };



    TestSVD();



    hipFree(d_voltage);
    hipFree(d_result);
    free(result);
    free(voltage);
    
}

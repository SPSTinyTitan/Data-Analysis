#include <matrix_wrappers.h>

namespace matrix{

    //C = (alpha * A) x (beta * B)
    //A - M x N
    //B - N x K
    //C - M x K
    __host__ void MultMat(float* A, float* B, float* C, float alpha, float beta, int M, int N, int K){

        //Initializing CuBlas
        hipblasHandle_t cublasH = NULL;
        hipblasStatus_t cublas_status = HIPBLAS_STATUS_SUCCESS;
        cublas_status = hipblasCreate(&cublasH);
        assert(HIPBLAS_STATUS_SUCCESS == cublas_status);

        //Zeroing results
        hipMemset(C, 0, M * K * sizeof(float));

        //Multiplying
        cublas_status = hipblasSgemm(
            cublasH,
            HIPBLAS_OP_N, HIPBLAS_OP_N,
            M, K, N,
            &alpha,
            A, M,
            B, N,
            &beta,
            C, M
        );
        assert(HIPBLAS_STATUS_SUCCESS == cublas_status);
    }

    //Calculates diagonal matrix multiplication
    //C = A x B
    //A - 1 x N diagonal (representing MxN)
    //B - N x K
    __host__ void MultMatD(float* A, float* B, float* C, int M, int N){
        
        //Initializing CuBlas
        hipblasHandle_t cublasH = NULL;
        hipblasStatus_t cublas_status = HIPBLAS_STATUS_SUCCESS;
        cublas_status = hipblasCreate(&cublasH);
        assert(HIPBLAS_STATUS_SUCCESS == cublas_status);

        //Zeroing results
        gpuErrchk(hipMemset(C, 0, M * N * sizeof(float)));

        //Multiplying
        cublas_status = hipblasSdgmm(
            cublasH, HIPBLAS_SIDE_LEFT,
            M, N,
            B, M,
            A, 1,
            C, M
        );
        assert(HIPBLAS_STATUS_SUCCESS == cublas_status);
    }

}
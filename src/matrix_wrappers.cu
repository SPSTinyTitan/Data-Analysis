#include "hip/hip_runtime.h"
#include "matrix_wrappers.h"

namespace matrix{
    
    //C = (alpha * A) x (beta * B)
    //A - M x N
    //B - N x K
    //C - M x K
    __host__ void mult(float* A, float* B, float* C, float alpha, float beta, bool TA, bool TB, int M, int N, int K){

        //Initializing CuBlas
        hipblasHandle_t cublasH = NULL;
        hipblasStatus_t cublas_status = HIPBLAS_STATUS_SUCCESS;
        hipblasOperation_t OA = (TA ? HIPBLAS_OP_T : HIPBLAS_OP_N);
        hipblasOperation_t OB = (TB ? HIPBLAS_OP_T : HIPBLAS_OP_N);
        cublas_status = hipblasCreate(&cublasH);
        assert(HIPBLAS_STATUS_SUCCESS == cublas_status);

        //Zeroing results
        hipMemset(C, 0, M * K * sizeof(float));

        //Multiplying
        cublas_status = hipblasSgemm(
            cublasH,
            OA, OB,
            M, K, N,
            &alpha,
            A, M,
            B, N,
            &beta,
            C, M
        );
        assert(HIPBLAS_STATUS_SUCCESS == cublas_status);
        gpuErrchk(hipDeviceSynchronize());
        if (cublasH) hipblasDestroy(cublasH);
    }

    //Calculates diagonal matrix multiplication
    //C = A x B
    //A - 1 x N diagonal (representing MxN)
    //B - N x K
    __host__ void multD(float* A, float* B, float* C, int M, int N){
        
        //Initializing CuBlas
        hipblasHandle_t cublasH = NULL;
        hipblasStatus_t cublas_status = HIPBLAS_STATUS_SUCCESS;
        cublas_status = hipblasCreate(&cublasH);
        assert(HIPBLAS_STATUS_SUCCESS == cublas_status);

        //Zeroing results
        gpuErrchk(hipMemset(C, 0, M * N * sizeof(float)));

        //Multiplying
        cublas_status = hipblasSdgmm(
            cublasH, HIPBLAS_SIDE_LEFT,
            M, N,
            B, M,
            A, 1,
            C, M
        );
        assert(HIPBLAS_STATUS_SUCCESS == cublas_status);
        gpuErrchk(hipDeviceSynchronize());
        if (cublasH ) hipblasDestroy(cublasH);
    }

    //Nvidia Reference implementation
    __global__ void transpose_(float *A, float *B, int M, int N){
        __shared__ float block[BLOCK_DIM][BLOCK_DIM+1];
        
        // read the matrix tile into shared memory
        unsigned int xIndex = blockIdx.x * BLOCK_DIM + threadIdx.x;
        unsigned int yIndex = blockIdx.y * BLOCK_DIM + threadIdx.y;
        if((xIndex < M) && (yIndex < N))
        {
            unsigned int index_in = yIndex * M + xIndex;
            block[threadIdx.y][threadIdx.x] = A[index_in];
        }

        __syncthreads();

        // write the transposed matrix tile to global memory
        xIndex = blockIdx.y * BLOCK_DIM + threadIdx.x;
        yIndex = blockIdx.x * BLOCK_DIM + threadIdx.y;
        if((xIndex < N) && (yIndex < M))
        {
            unsigned int index_out = yIndex * N + xIndex;
            B[index_out] = block[threadIdx.x][threadIdx.y];
        }
    }
    __host__ void transpose(float* A, float* B, int M, int N){

        float* C;
        //Copy matrix if storing back
        if (A == B){
            gpuErrchk(hipMalloc(&C, M * N * sizeof(float)));
            vector::copy(C, A, M * N);
        }
        else
            C = A;
        dim3 grid((M - 1 + BLOCK_DIM) / BLOCK_DIM, (N - 1 + BLOCK_DIM) / BLOCK_DIM, 1);
        dim3 threads(BLOCK_DIM, BLOCK_DIM, 1);
        transpose_<<<grid, threads>>>(C, B, M, N);
        gpuErrchk(hipDeviceSynchronize());
        if (A == B && C) hipFree(C);
    }

    __host__ void inverse (float* A, float* B, int N){
        float* C;
        if (A == B)
            hipMalloc(&C, N * N * sizeof(float));
        else C = B;

        //Initializing CuBlas
        hipsolverHandle_t cusolverH = NULL;
        hipsolverStatus_t cusolver_status = HIPSOLVER_STATUS_SUCCESS;
        cusolver_status = hipsolverDnCreate(&cusolverH);
        assert(HIPSOLVER_STATUS_SUCCESS == cusolver_status);

        int lwork;

        cusolver_status = hipsolverDnSgetrf_bufferSize(
            cusolverH,
            N, N,
            A, N,
            &lwork 
        );
        gpuErrchk(hipDeviceSynchronize());
        assert(HIPSOLVER_STATUS_SUCCESS == cusolver_status);

        float* work;    gpuErrchk(hipMalloc(&work, lwork * sizeof(float)));
        int* P;         gpuErrchk(hipMalloc(&P, N * sizeof(int)));
        int* devInfo;   gpuErrchk(hipMalloc(&devInfo, sizeof(int))); 

        cusolver_status = hipsolverDnSgetrf(
            cusolverH,
            N, N,
            A, N,
            work,
            P,
            devInfo
        );
        gpuErrchk(hipDeviceSynchronize());
        assert(HIPSOLVER_STATUS_SUCCESS == cusolver_status);


        float* identity = (float*)malloc(N * N * sizeof(float));
        for (int i = 0; i < N * N; i++)
            identity[i] = 0;
        for (int i = 0; i < N; i++)
            identity[i * (N+1)] = 1; 
        gpuErrchk(hipMemcpy(C, identity, N * N * sizeof(float), hipMemcpyHostToDevice));
        hipsolverDnSgetrs(cusolverH,
            HIPBLAS_OP_N,
            N, N,
            A, N,
            P,
            C, N,
            devInfo 
        );
        gpuErrchk(hipDeviceSynchronize());
        assert(HIPSOLVER_STATUS_SUCCESS == cusolver_status);

        if (identity)   free(identity);
        if (devInfo)    hipFree(devInfo);
        if (work)       hipFree(work);
        if (P)          hipFree(P);  
        if (A == B){
            vector::copy(A, C, N * N);
            hipFree(C);
        }
    }
}